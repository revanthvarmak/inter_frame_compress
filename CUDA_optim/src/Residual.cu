#include "hip/hip_runtime.h"
#include "Residual.h"
#include "constants.h"
#include <hip/hip_runtime.h>
#include <vector>

// Kernels
__global__ void kResidual(const uint8_t*cur,const uint8_t*ref, const MotionVector* MV, int W,int H, int C, uint8_t*res) {
    // Calculate pixel coordinates in the image
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x >= W || y >= H) return;

    // Determine the macroblock that this pixel is in and retrieve the motion vector
    int mbX = x / BLOCK_SIZE;
    int mbY = y / BLOCK_SIZE;

    // row major order calculatiojn
    int mbsX= (W + BLOCK_SIZE - 1) / BLOCK_SIZE;
    MotionVector mv = MV[mbY * mbsX + mbX];

    // Determine the reference pixel coordinates using the motion vectors
    int refX = min(max(x + mv.dx,0), W-1);
    int refY = min(max(y + mv.dy,0), H-1);

    for (int c = 0; c < C; c++) {
        int pixelIdx = (y * W + x) * C + c;
        int refIdx = (refY * W + refX) * C + c;
        
        int diff = int(cur[pixelIdx]) - int(ref[refIdx]);
        diff = max(-127, min(127, diff)); 
        res[pixelIdx] = uint8_t(diff + RESIDUAL_OFFSET);
    }
}

__global__ void kReconstruct(const uint8_t*ref,const uint8_t*res, const MotionVector* MV, int W,int H,int C, uint8_t*out) {
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x >= W || y >= H) return;

    int mbX = x / BLOCK_SIZE;
    int mbY = y / BLOCK_SIZE;
    int mbsX= (W + BLOCK_SIZE - 1) / BLOCK_SIZE;
    MotionVector mv = MV[mbY * mbsX + mbX];

    int refX = min(max(x + mv.dx,0), W-1);
    int refY = min(max(y + mv.dy,0), H-1);

    for (int c = 0; c < C; c++) {
        int pixelIdx = (y * W + x) * C + c;
        int refIdx = (refY * W + refX) * C + c;
        
        int pix = int(ref[refIdx]) + int(res[pixelIdx]) - RESIDUAL_OFFSET;
        pix = pix < 0 ? 0 : (pix > 255 ? 255 : pix);
        out[pixelIdx] = uint8_t(pix);
    }
}

// Host functions

// Calculate number of threads and blocks requried to cover the entire 2D frame
void launch2D(dim3 &grid, dim3 &block, int W, int H){
    // Since each warp of 32 threads execute together, and better performance is achieved if all threads in a warp access consecutive memory,
    // x dimension of block should be multiple of 32

    block = dim3(32, 16);  
    grid  = dim3((W+block.x-1)/block.x, (H+block.y-1)/block.y);
}

Frame calculateResidual(const Frame& cur,const Frame& ref, const std::vector<MotionVector>& MV) {
    int W = cur.width;
    int H = cur.height;
    int C = cur.channels;

    // Create CUDA stream
    hipStream_t stream;
    hipStreamCreate(&stream);

    Frame residual(W, H, P_frame, C);

    // device buffers
    uint8_t *dCur, *dRef, *dRes; 
    MotionVector* dMV;

    size_t frameBytes = size_t(W) * H * C;

    hipMalloc(&dCur,frameBytes);
    hipMalloc(&dRef,frameBytes);
    hipMalloc(&dRes,frameBytes);
    hipMalloc(&dMV, MV.size() * sizeof(MotionVector));

    hipMemcpyAsync(dCur, cur.data, frameBytes, hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(dRef, ref.data, frameBytes, hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(dMV, MV.data(), MV.size() * sizeof(MotionVector), hipMemcpyHostToDevice, stream);

    dim3 grid, block; 
    launch2D(grid, block, W, H);
    kResidual<<<grid, block, 0, stream>>>(dCur, dRef, dMV, W, H, C, dRes);
    hipMemcpyAsync(residual.data,dRes,frameBytes,hipMemcpyDeviceToHost,stream);


    // Cleanup CUDA memory
    hipStreamSynchronize(stream);
    hipFree(dCur); 
    hipFree(dRef); 
    hipFree(dRes); 
    hipFree(dMV);
    hipStreamDestroy(stream);
    return residual;
}

Frame decodeP_Frame(const Frame& ref, const std::vector<MotionVector>& MV, const Frame& residual) {
    int W = ref.width;
    int H = ref.height;
    int C = ref.channels;
    Frame out(W, H, P_frame, C);

    // Create CUDA stream
    hipStream_t stream;
    hipStreamCreate(&stream);

    uint8_t *dRef,*dRes,*dOut; 
    MotionVector* dMV;
    size_t frameBytes = size_t(W) * H * C;
    hipMalloc(&dRef,frameBytes);
    hipMalloc(&dRes,frameBytes);
    hipMalloc(&dOut,frameBytes);
    hipMalloc(&dMV ,MV.size() * sizeof(MotionVector));

    hipMemcpyAsync(dRef, ref.data, frameBytes, hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(dRes, residual.data, frameBytes, hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(dMV, MV.data(), MV.size() * sizeof(MotionVector), hipMemcpyHostToDevice, stream);

    dim3 grid,block; 
    launch2D(grid, block, W, H);
    kReconstruct<<<grid,block,0,stream>>>(dRef, dRes, dMV, W, H, C, dOut);
    
    hipMemcpyAsync(out.data,dOut,frameBytes,hipMemcpyDeviceToHost,stream);


    hipStreamSynchronize(stream);
    hipFree(dRef); 
    hipFree(dRes); 
    hipFree(dOut); 
    hipFree(dMV);
    hipStreamDestroy(stream);
    return out;
}
